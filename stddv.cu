
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

// setting the default values, change them per requirment.
// TODO: need to stress test if grid dimension occurs
#define N 10
const int threadsPerBlock = 256;
const int blocksPerGrid = min(32, (N + threadsPerBlock-1)/threadsPerBlock);

__global__
void stddv(float *a, float* mean, float* stddv){
    __shared__ float cache_mean[threadsPerBlock];
    __shared__ float cache_var[threadsPerBlock];
    __shared__ int cache_count[threadsPerBlock];

    int tid = threadIdx.x + blockIdx.x*blockDim.x;
    int cacheIndex = threadIdx.x;

    float tMean = 0.0f;
    float tVar = 0.0f;
    int cnt = 1;

    while(tid < N){
        float x = a[tid];
        float oldMean = tMean;
        tMean = tMean + (x-tMean)/(cnt);
        tVar = tVar + (x-tMean)*(x-oldMean);
        cnt = cnt + 1;
        tid += blockDim.x * gridDim.x;
    }

    cache_mean[cacheIndex] = tMean;
    cache_var[cacheIndex] = tVar;
    cache_count[cacheIndex] = cnt-1;
    __syncthreads();

    int total_i = min(blockDim.x, N-blockDim.x*blockIdx.x);
    int i = (total_i==1) ? total_i/2 : (total_i+1)/2;

    while(i != 0){
        if(cacheIndex < i){
            int idx1 = cacheIndex; int idx2 = cacheIndex + i;
            int n1 = cache_count[idx1]; int n2 = cache_count[idx2];
            float mean1 = cache_mean[idx1]; float mean2 = cache_mean[idx2];
            float var1 = cache_var[idx1]; float var2 = cache_var[idx2];
            if(!(total_i%2 && cacheIndex == i-1)){
                int n = n1 + n2;
                float delta = mean2 - mean1;
                float combined_mean = mean1 + (delta*((float)n2/(float)n));
                float combined_var = var2 + var1 + ((delta*delta)*(float)n1*(float)n2/(float)n);
                cache_count[idx1] = n;
                cache_mean[idx1] = combined_mean;
                cache_var[idx1] = combined_var;
            }
        }
        __syncthreads();
        total_i = i;
        i = (i==1) ? i/2 : (i+1)/2;
    }

    if(cacheIndex == 0){
        mean[blockIdx.x] = cache_mean[0];
        int size = min(blockDim.x, N-blockDim.x*blockIdx.x);
        stddv[blockIdx.x] = cache_var[0];
    }
    __syncthreads();

    // TODO: parallelize this part.
    // computing mean and variance across
    // the block in a parallel way
    if(blockIdx.x==0 && cacheIndex==0){
        int n1 = min(blockDim.x, N-blockDim.x*blockIdx.x);
        float mean1 = mean[0];
        float var1 = stddv[0];
        for(int i=1; i<gridDim.x; i++){
            float mean2 = mean[i];
            float var2 = stddv[i];
            int n2 = min(blockDim.x, N-blockDim.x*i);
            int n = n1 + n2;
            float delta = mean2 - mean1;
            float combined_mean = mean1 + (delta*((float)n2/(float)n));
            float combined_var = var2 + var1 + ((delta*delta)*(float)n1*(float)n2/(float)n);
            mean1 = combined_mean;
            var1 = combined_var;
            n1 = n;
        }
        mean[0] = mean1;
        stddv[0] = sqrt(var1/(float)(n1-1));
    }

    return;
}

int main(void){
    int output_size = blocksPerGrid;
    float a[N], b[output_size], c[output_size];
    float *dev_a, *dev_b, *dev_c;

    hipMalloc((void**)&dev_a, sizeof(float)*N);
    hipMalloc((void**)&dev_b, sizeof(float)*output_size);
    hipMalloc((void**)&dev_c, sizeof(float)*output_size);

    for(int i=0; i<N; i++){
        a[i] = (float)i;
    }

    hipMemcpy(dev_a, a, sizeof(float)*N, hipMemcpyHostToDevice);

    //call the kernel
    printf("bpg, tpb = (%d, %d)\n", blocksPerGrid, threadsPerBlock);
    stddv<<<blocksPerGrid, threadsPerBlock>>> (dev_a, dev_b, dev_c);


    hipMemcpy(b, dev_b, sizeof(float)*output_size, hipMemcpyDeviceToHost);
    hipMemcpy(c, dev_c, sizeof(float)*output_size, hipMemcpyDeviceToHost);
    printf("mean (a.k.a b[0]) = %f\n", b[0]);
    printf("stddv (a.k.a c[0]) = %f\n", c[0]);


    hipFree(dev_a); 
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}